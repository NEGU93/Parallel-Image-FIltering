#include "hip/hip_runtime.h"
#include "cudaKernel.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void apply_blur_kernel(int height, int width, int size, pixel * p, pixel * newp, int threshold) {
	int j, k;
	int row = blockDim.x * blockIdx.x + threadIdx.x;
	int	col = blockDim.y * blockIdx.y + threadIdx.y;
	//cuPrintf("j = %d + %d < %d. j+= %d\n", row, size, height / 10 - size, blockDim.y );
	//cuPrintf("k = %d + %d < %d. k+= %d\n", col, size, width - size, blockDim.x);
	for (j = row + size; j < height - size; j += blockDim.x * gridDim.x ) {
		for (k = col + size; k < width - size; k += blockDim.y * gridDim.y ) {
			int stencil_j, stencil_k;
			int t = 0;

			for (stencil_j = -size; stencil_j <= size; stencil_j++) {
				for (stencil_k = -size; stencil_k <= size; stencil_k++) {
					switch(threadIdx.z) {
						case 0:
							t += p[CONV(j + stencil_j, k + stencil_k, width)].r;
							break;
						case 1:
							t += p[CONV(j + stencil_j, k + stencil_k, width)].g;
							break;
						case 2:
							t += p[CONV(j + stencil_j, k + stencil_k, width)].b;
							break;
					}
				}
			}

			switch(threadIdx.z) {
				case 0:
					newp[CONV(j, k, width)].r = t / ((2 * size + 1)*(2 * size + 1));
					break;
				case 1:
					newp[CONV(j, k, width)].g = t / ((2 * size + 1)*(2 * size + 1));
					break;
				case 2:
					newp[CONV(j, k, width)].b = t / ((2 * size + 1)*(2 * size + 1));
					break;
			}
		}
	}
}
__global__ void apply_kernel(int height, int width, int size, pixel * myImg, pixel * newp, int threshold, int *end) {
	int j, k;
	int row = blockDim.x * blockIdx.x + threadIdx.x;
	int	col = blockDim.y * blockIdx.y + threadIdx.y;	

	for (j = row + size; j < height - size; j += blockDim.x * gridDim.x ) {
		for (k = col + size; k < width - size; k += blockDim.y * gridDim.y ) {
		    float diff;

			switch(threadIdx.z) {
				case 0:
					diff = newp[CONV(j,k,width)].r - myImg[CONV(j,k,width)].r;
					break;
				case 1:
					diff = newp[CONV(j,k,width)].g - myImg[CONV(j,k,width)].g;
					break;
				case 2:
					diff = newp[CONV(j,k,width)].b - myImg[CONV(j,k,width)].b;
					break;
			}

		    if ( diff > threshold || -diff > threshold ) { * end = 0; }

			switch(threadIdx.z) {
				case 0:
					myImg[CONV(j,k,width)].r = newp[CONV(j,k,width)].r;
					break;
				case 1:
					myImg[CONV(j,k,width)].g = newp[CONV(j,k,width)].g;
					break;
				case 2:
					myImg[CONV(j,k,width)].b = newp[CONV(j,k,width)].b;
					break;
			}
		}
	}
}

/* Cuda inits */
void alloc_device_pixel_array(int w, int h, pixel **p) {
	hipError_t err = hipMalloc(p, w * h * sizeof(pixel));
	if (err != hipSuccess) { 
		fprintf(stderr, "GPUassert: %s in function %s line %d\n", hipGetErrorString(err), __FUNCTION__, __LINE__);
		abort(); 
	}
}
void transfer_pixel_array_H2D(int N, pixel *p, pixel *d_p) {
	hipError_t err = hipMemcpy(d_p, p, N * sizeof(pixel), hipMemcpyHostToDevice);
	if (err != hipSuccess) { 
		fprintf(stderr, "GPUassert: %s in function %s line %d\n", hipGetErrorString(err), __FUNCTION__, __LINE__);
		abort(); 
	}
}
void transfer_pixel_array_D2H(int N, pixel *p, pixel *d_p) {
	hipError_t err = hipMemcpy(p, d_p, N * sizeof(pixel), hipMemcpyDeviceToHost);
	if (err != hipSuccess) { 
		fprintf(stderr, "GPUassert: %s in function %s line %d\n", hipGetErrorString(err), __FUNCTION__, __LINE__);
		abort(); 
	}
}

int apply_blur_cuda(int height, int width, int size, pixel * p, int threshold) {
	pixel * d_p;
	pixel * d_new;
	int end = 1;
	int * d_end;
	dim3 gridDim(2, 2, 1);
	dim3 blockDim(8, 8, 3 /* Here it MUST be 3 */);
	/* Alloc everything in device */
	alloc_device_pixel_array(width, height, &d_p);
	alloc_device_pixel_array(width, height, &d_new);
	hipMalloc((void**)&d_end, sizeof(int));
	//alloc_device_int(&end);
	/* Copy to memory */
	transfer_pixel_array_H2D(width*height, p, d_p);
	hipMemcpy(d_end, &end, sizeof(int), hipMemcpyHostToDevice);
	/* Call Kernel */
	//cudaPrintfInit();
	apply_blur_kernel<<<gridDim, blockDim>>>(height, width, size, d_p, d_new, threshold);
	apply_kernel<<<gridDim, blockDim>>>(height, width, size, d_p, d_new, threshold, d_end);
	//cudaPrintfDisplay(stdout, true);
    //cudaPrintfEnd();
	/* Copy the result */
	transfer_pixel_array_D2H(width*height, p, d_p);
	hipMemcpy(&end, d_end, sizeof(int), hipMemcpyDeviceToHost);
	/* Free everything in the device */
	hipFree(d_p);
	hipFree(d_new);	

	return end;
}


