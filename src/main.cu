#include "hip/hip_runtime.h"
/*
 * INF560
 *
 * Image Filtering Project
 */

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <math.h>
#ifdef _WIN32
#include <windows.h>					// for Windows APIs
#include ""	// For VS to recognize the blockIdx
#else
#include <sys/time.h>
#endif

#include <cuPrintf.cu>
#include "gif_lib.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define SOBELF_DEBUG 0

#define CONV(l,c,nb_c) \
    (l)*(nb_c)+(c)

/* Represent one pixel from the image */
typedef struct pixel {
    int r ; /* Red */
    int g ; /* Green */
    int b ; /* Blue */
} pixel ;

/* Represent one GIF image (animated or not */
typedef struct animated_gif {
    int n_images ; 		/* Number of images */
    int * width ; 		/* Width of each image */
    int * height ; 		/* Height of each image */
    pixel ** p ; 		/* Pixels of each image */
    GifFileType * g ; 	/* Internal representation.
                         DO NOT MODIFY */
} animated_gif ;

__global__ void apply_blur_top_kernel(int height, int width, int size, pixel * p, pixel * newp) {
	int j, k;
	int row = threadIdx.x;
	int	col = threadIdx.y;
	//cuPrintf("j = %d + %d < %d. j+= %d\n", row, size, height / 10 - size, blockDim.y );
	//cuPrintf("k = %d + %d < %d. k+= %d\n", col, size, width - size, blockDim.x);
	for (j = row + size; j < height / 10 - size; j += blockDim.x ) {
		for (k = col + size; k < width - size; k += blockDim.y ) {
			int stencil_j, stencil_k;
			int t_r = 0;
			int t_g = 0;
			int t_b = 0;

			for (stencil_j = -size; stencil_j <= size; stencil_j++) {
				for (stencil_k = -size; stencil_k <= size; stencil_k++) {
					t_r += p[CONV(j + stencil_j, k + stencil_k, width)].r;
					t_g += p[CONV(j + stencil_j, k + stencil_k, width)].g;
					t_b += p[CONV(j + stencil_j, k + stencil_k, width)].b;
				}
			}

			newp[CONV(j, k, width)].r = t_r / ((2 * size + 1)*(2 * size + 1));
			newp[CONV(j, k, width)].g = t_g / ((2 * size + 1)*(2 * size + 1));
			newp[CONV(j, k, width)].b = t_b / ((2 * size + 1)*(2 * size + 1));
		}
	}
}

/* Cuda inits */
void alloc_device_pixel_array(int w, int h, pixel **p) {
	hipError_t err = hipMalloc(p, w * h * sizeof(pixel));
	if (err != hipSuccess) { 
		fprintf(stderr, "GPUassert: %s in function %s line %d\n", hipGetErrorString(err), __FUNCTION__, __LINE__);
		abort(); 
	}
}
void transfer_pixel_array_H2D(int N, pixel *p, pixel *d_p) {
	hipError_t err = hipMemcpy(d_p, p, N * sizeof(pixel), hipMemcpyHostToDevice);
	if (err != hipSuccess) { 
		fprintf(stderr, "GPUassert: %s in function %s line %d\n", hipGetErrorString(err), __FUNCTION__, __LINE__);
		abort(); 
	}
}
void transfer_pixel_array_D2H(int N, pixel *p, pixel *d_p) {
	hipError_t err = hipMemcpy(p, d_p, N * sizeof(pixel), hipMemcpyDeviceToHost);
	if (err != hipSuccess) { 
		fprintf(stderr, "GPUassert: %s in function %s line %d\n", hipGetErrorString(err), __FUNCTION__, __LINE__);
		abort(); 
	}
}
void apply_blur_top(int height, int width, int size, pixel * p, pixel * newp) {
	pixel * d_p;
	pixel * d_new;
	dim3 gridDim(1);
	dim3 blockDim(8, 8);
	/* Alloc everything in device */
	alloc_device_pixel_array(width, height, &d_p);
	alloc_device_pixel_array(width, height, &d_new);
	/* Copy to memory */
	transfer_pixel_array_H2D(width*height, p, d_p);
	/* Call Kernel */
	//cudaPrintfInit();
	apply_blur_top_kernel<<<gridDim, blockDim>>>(height, width, size, d_p, d_new);
	//cudaPrintfDisplay(stdout, true);
    //cudaPrintfEnd();
	/* Copy the result */
	transfer_pixel_array_D2H(width*height, newp, d_new);
	/* Free everything in the device */
	hipFree(d_p);
	hipFree(d_new);
}
/*
 * Load a GIF image from a file and return a
 * structure of type animated_gif.
 */
animated_gif * load_pixels( char * filename ) {
    GifFileType * g ;
    ColorMapObject * colmap ;
    int error ;
    int n_images ;
    int * width ;
    int * height ;
    pixel ** p ;
    int i ;
    animated_gif * image ;

    /* Open the GIF image (read mode) */
    g = DGifOpenFileName( filename, &error ) ;
    if ( g == NULL ) 
    {
        fprintf( stderr, "Error DGifOpenFileName %s\n", filename ) ;
        return NULL ;
    }

    /* Read the GIF image */
    error = DGifSlurp( g ) ;
    if ( error != GIF_OK ) {
        fprintf( stderr, 
                "Error DGifSlurp: %d <%s>\n", error, GifErrorString(g->Error) ) ;
        return NULL ;
    }

    /* Grab the number of images and the size of each image */
    n_images = g->ImageCount ;

    width = (int *)malloc( n_images * sizeof( int ) ) ;
    if ( width == NULL ) {
        fprintf( stderr, "Unable to allocate width of size %d\n",
                n_images ) ;
        return 0 ;
    }

    height = (int *)malloc( n_images * sizeof( int ) ) ;
    if ( height == NULL ) {
        fprintf( stderr, "Unable to allocate height of size %d\n",
                n_images ) ;
        return 0 ;
    }

    /* Fill the width and height */
    for ( i = 0 ; i < n_images ; i++ ) {
        width[i] = g->SavedImages[i].ImageDesc.Width ;
        height[i] = g->SavedImages[i].ImageDesc.Height ;

#if SOBELF_DEBUG
        printf( "Image %d: l:%d t:%d w:%d h:%d interlace:%d localCM:%p\n",
                i, 
                g->SavedImages[i].ImageDesc.Left,
                g->SavedImages[i].ImageDesc.Top,
                g->SavedImages[i].ImageDesc.Width,
                g->SavedImages[i].ImageDesc.Height,
                g->SavedImages[i].ImageDesc.Interlace,
                g->SavedImages[i].ImageDesc.ColorMap
                ) ;
#endif
    }


    /* Get the global colormap */
    colmap = g->SColorMap ;
    if ( colmap == NULL ) {
        fprintf( stderr, "Error global colormap is NULL\n" ) ;
        return NULL ;
    }

#if SOBELF_DEBUG
    printf( "Global CM: count:%d bpp:%d sort:%d\n",
            g->SColorMap->ColorCount,
            g->SColorMap->BitsPerPixel,
            g->SColorMap->SortFlag
            ) ;
#endif

    /* Allocate the array of pixels to be returned */
    p = (pixel **)malloc( n_images * sizeof( pixel * ) ) ;
    if ( p == NULL ) {
        fprintf( stderr, "Unable to allocate array of %d images\n",
                n_images ) ;
        return NULL ;
    }

    for ( i = 0 ; i < n_images ; i++ ) {
        p[i] = (pixel *)malloc( width[i] * height[i] * sizeof( pixel ) ) ;
        if ( p[i] == NULL ) {
        fprintf( stderr, "Unable to allocate %d-th array of %d pixels\n", i, width[i] * height[i] ) ;
        return NULL ;
        }
    }
    
    /* Fill pixels */

    /* For each image */
    for ( i = 0 ; i < n_images ; i++ ) {
        int j ;

        /* Get the local colormap if needed */
        if ( g->SavedImages[i].ImageDesc.ColorMap ) {

            /* TODO No support for local color map */
            fprintf( stderr, "Error: application does not support local colormap\n" ) ;
            return NULL ;

            colmap = g->SavedImages[i].ImageDesc.ColorMap ;
        }

        /* Traverse the image and fill pixels */
        for ( j = 0 ; j < width[i] * height[i] ; j++ ) {
            int c ;

            c = g->SavedImages[i].RasterBits[j] ;

            p[i][j].r = colmap->Colors[c].Red ;
            p[i][j].g = colmap->Colors[c].Green ;
            p[i][j].b = colmap->Colors[c].Blue ;
        }
    }

    /* Allocate image info */
    image = (animated_gif *)malloc( sizeof(animated_gif) ) ;
    if ( image == NULL ) 
    {
        fprintf( stderr, "Unable to allocate memory for animated_gif\n" ) ;
        return NULL ;
    }

    /* Fill image fields */
    image->n_images = n_images ;
    image->width = width ;
    image->height = height ;
    image->p = p ;
    image->g = g ;

#if SOBELF_DEBUG
    printf( "-> GIF w/ %d image(s) with first image of size %d x %d\n",
            image->n_images, image->width[0], image->height[0] ) ;
#endif

    return image ;
}

int output_modified_read_gif( char * filename, GifFileType * g ) {
    GifFileType * g2 ;
    int error2 ;

#if SOBELF_DEBUG
    printf( "Starting output to file %s\n", filename ) ;
#endif

    g2 = EGifOpenFileName( filename, false, &error2 ) ;
    if ( g2 == NULL )
    {
        fprintf( stderr, "Error EGifOpenFileName %s\n",
                filename ) ;
        return 0 ;
    }

    g2->SWidth = g->SWidth ;
    g2->SHeight = g->SHeight ;
    g2->SColorResolution = g->SColorResolution ;
    g2->SBackGroundColor = g->SBackGroundColor ;
    g2->AspectByte = g->AspectByte ;
    g2->SColorMap = g->SColorMap ;
    g2->ImageCount = g->ImageCount ;
    g2->SavedImages = g->SavedImages ;
    g2->ExtensionBlockCount = g->ExtensionBlockCount ;
    g2->ExtensionBlocks = g->ExtensionBlocks ;

    error2 = EGifSpew( g2 ) ;
    if ( error2 != GIF_OK ) 
    {
        fprintf( stderr, "Error after writing g2: %d <%s>\n", 
                error2, GifErrorString(g2->Error) ) ;
        return 0 ;
    }

    return 1 ;
}
int store_pixels( char * filename, animated_gif * image ) {
    int n_colors = 0 ;
    pixel ** p ;
    int i, j, k ;
    GifColorType * colormap ;

    /* Initialize the new set of colors */
    colormap = (GifColorType *)malloc( 256 * sizeof( GifColorType ) ) ;
    if ( colormap == NULL ) 
    {
        fprintf( stderr,
                "Unable to allocate 256 colors\n" ) ;
        return 0 ;
    }

    /* Everything is white by default */
    for ( i = 0 ; i < 256 ; i++ ) 
    {
        colormap[i].Red = 255 ;
        colormap[i].Green = 255 ;
        colormap[i].Blue = 255 ;
    }

    /* Change the background color and store it */
    int moy ;
    moy = (
            image->g->SColorMap->Colors[ image->g->SBackGroundColor ].Red
            +
            image->g->SColorMap->Colors[ image->g->SBackGroundColor ].Green
            +
            image->g->SColorMap->Colors[ image->g->SBackGroundColor ].Blue
            )/3 ;
    if ( moy < 0 ) moy = 0 ;
    if ( moy > 255 ) moy = 255 ;

#if SOBELF_DEBUG
    printf( "[DEBUG] Background color (%d,%d,%d) -> (%d,%d,%d)\n",
            image->g->SColorMap->Colors[ image->g->SBackGroundColor ].Red,
            image->g->SColorMap->Colors[ image->g->SBackGroundColor ].Green,
            image->g->SColorMap->Colors[ image->g->SBackGroundColor ].Blue,
            moy, moy, moy ) ;
#endif

    colormap[0].Red = moy ;
    colormap[0].Green = moy ;
    colormap[0].Blue = moy ;

    image->g->SBackGroundColor = 0 ;

    n_colors++ ;

    /* Process extension blocks in main structure */
    for ( j = 0 ; j < image->g->ExtensionBlockCount ; j++ )
    {
        int f ;

        f = image->g->ExtensionBlocks[j].Function ;
        if ( f == GRAPHICS_EXT_FUNC_CODE )
        {
            int tr_color = image->g->ExtensionBlocks[j].Bytes[3] ;

            if ( tr_color >= 0 &&
                    tr_color < 255 )
            {

                int found = -1 ;

                moy = 
                    (
                     image->g->SColorMap->Colors[ tr_color ].Red
                     +
                     image->g->SColorMap->Colors[ tr_color ].Green
                     +
                     image->g->SColorMap->Colors[ tr_color ].Blue
                    ) / 3 ;
                if ( moy < 0 ) moy = 0 ;
                if ( moy > 255 ) moy = 255 ;

#if SOBELF_DEBUG
                printf( "[DEBUG] Transparency color image %d (%d,%d,%d) -> (%d,%d,%d)\n",
                        i,
                        image->g->SColorMap->Colors[ tr_color ].Red,
                        image->g->SColorMap->Colors[ tr_color ].Green,
                        image->g->SColorMap->Colors[ tr_color ].Blue,
                        moy, moy, moy ) ;
#endif

                for ( k = 0 ; k < n_colors ; k++ )
                {
                    if ( 
                            moy == colormap[k].Red
                            &&
                            moy == colormap[k].Green
                            &&
                            moy == colormap[k].Blue
                       )
                    {
                        found = k ;
                    }
                }
                if ( found == -1  ) 
                {
                    if ( n_colors >= 256 ) 
                    {
                        fprintf( stderr, 
                                "Error: Found too many colors inside the image\n"
                               ) ;
                        return 0 ;
                    }

#if SOBELF_DEBUG
                    printf( "[DEBUG]\tNew color %d\n",
                            n_colors ) ;
#endif

                    colormap[n_colors].Red = moy ;
                    colormap[n_colors].Green = moy ;
                    colormap[n_colors].Blue = moy ;


                    image->g->ExtensionBlocks[j].Bytes[3] = n_colors ;

                    n_colors++ ;
                } else
                {
#if SOBELF_DEBUG
                    printf( "[DEBUG]\tFound existing color %d\n",
                            found ) ;
#endif
                    image->g->ExtensionBlocks[j].Bytes[3] = found ;
                }
            }
        }
    }

    for ( i = 0 ; i < image->n_images ; i++ )
    {
        for ( j = 0 ; j < image->g->SavedImages[i].ExtensionBlockCount ; j++ )
        {
            int f ;

            f = image->g->SavedImages[i].ExtensionBlocks[j].Function ;
            if ( f == GRAPHICS_EXT_FUNC_CODE )
            {
                int tr_color = image->g->SavedImages[i].ExtensionBlocks[j].Bytes[3] ;

                if ( tr_color >= 0 &&
                        tr_color < 255 )
                {

                    int found = -1 ;

                    moy = 
                        (
                         image->g->SColorMap->Colors[ tr_color ].Red
                         +
                         image->g->SColorMap->Colors[ tr_color ].Green
                         +
                         image->g->SColorMap->Colors[ tr_color ].Blue
                        ) / 3 ;
                    if ( moy < 0 ) moy = 0 ;
                    if ( moy > 255 ) moy = 255 ;

#if SOBELF_DEBUG
                    printf( "[DEBUG] Transparency color image %d (%d,%d,%d) -> (%d,%d,%d)\n",
                            i,
                            image->g->SColorMap->Colors[ tr_color ].Red,
                            image->g->SColorMap->Colors[ tr_color ].Green,
                            image->g->SColorMap->Colors[ tr_color ].Blue,
                            moy, moy, moy ) ;
#endif

                    for ( k = 0 ; k < n_colors ; k++ )
                    {
                        if ( 
                                moy == colormap[k].Red
                                &&
                                moy == colormap[k].Green
                                &&
                                moy == colormap[k].Blue
                           )
                        {
                            found = k ;
                        }
                    }
                    if ( found == -1  ) 
                    {
                        if ( n_colors >= 256 ) 
                        {
                            fprintf( stderr, 
                                    "Error: Found too many colors inside the image\n"
                                   ) ;
                            return 0 ;
                        }

#if SOBELF_DEBUG
                        printf( "[DEBUG]\tNew color %d\n",
                                n_colors ) ;
#endif

                        colormap[n_colors].Red = moy ;
                        colormap[n_colors].Green = moy ;
                        colormap[n_colors].Blue = moy ;


                        image->g->SavedImages[i].ExtensionBlocks[j].Bytes[3] = n_colors ;

                        n_colors++ ;
                    } else
                    {
#if SOBELF_DEBUG
                        printf( "[DEBUG]\tFound existing color %d\n",
                                found ) ;
#endif
                        image->g->SavedImages[i].ExtensionBlocks[j].Bytes[3] = found ;
                    }
                }
            }
        }
    }

#if SOBELF_DEBUG
    printf( "[DEBUG] Number of colors after background and transparency: %d\n",
            n_colors ) ;
#endif

    p = image->p ;

    /* Find the number of colors inside the image */
    for ( i = 0 ; i < image->n_images ; i++ )
    {

#if SOBELF_DEBUG
        printf( "OUTPUT: Processing image %d (total of %d images) -> %d x %d\n",
                i, image->n_images, image->width[i], image->height[i] ) ;
#endif

        for ( j = 0 ; j < image->width[i] * image->height[i] ; j++ ) 
        {
            int found = 0 ;
            for ( k = 0 ; k < n_colors ; k++ )
            {
                if ( p[i][j].r == colormap[k].Red &&
                        p[i][j].g == colormap[k].Green &&
                        p[i][j].b == colormap[k].Blue )
                {
                    found = 1 ;
                }
            }

            if ( found == 0 ) 
            {
                if ( n_colors >= 256 ) 
                {
                    fprintf( stderr, 
                            "Error: Found too many colors inside the image\n"
                           ) ;
                    return 0 ;
                }

#if SOBELF_DEBUG
                printf( "[DEBUG] Found new %d color (%d,%d,%d)\n",
                        n_colors, p[i][j].r, p[i][j].g, p[i][j].b ) ;
#endif

                colormap[n_colors].Red = p[i][j].r ;
                colormap[n_colors].Green = p[i][j].g ;
                colormap[n_colors].Blue = p[i][j].b ;
                n_colors++ ;
            }
        }
    }

#if SOBELF_DEBUG
    printf( "OUTPUT: found %d color(s)\n", n_colors ) ;
#endif


    /* Round up to a power of 2 */
    if ( n_colors != (1 << GifBitSize(n_colors) ) )
    {
        n_colors = (1 << GifBitSize(n_colors) ) ;
    }

#if SOBELF_DEBUG
    printf( "OUTPUT: Rounding up to %d color(s)\n", n_colors ) ;
#endif

    /* Change the color map inside the animated gif */
    ColorMapObject * cmo ;

    cmo = GifMakeMapObject( n_colors, colormap ) ;
    if ( cmo == NULL )
    {
        fprintf( stderr, "Error while creating a ColorMapObject w/ %d color(s)\n",
                n_colors ) ;
        return 0 ;
    }

    image->g->SColorMap = cmo ;

    /* Update the raster bits according to color map */
    for ( i = 0 ; i < image->n_images ; i++ )
    {
        for ( j = 0 ; j < image->width[i] * image->height[i] ; j++ ) 
        {
            int found_index = -1 ;
            for ( k = 0 ; k < n_colors ; k++ ) 
            {
                if ( p[i][j].r == image->g->SColorMap->Colors[k].Red &&
                        p[i][j].g == image->g->SColorMap->Colors[k].Green &&
                        p[i][j].b == image->g->SColorMap->Colors[k].Blue )
                {
                    found_index = k ;
                }
            }

            if ( found_index == -1 ) 
            {
                fprintf( stderr,
                        "Error: Unable to find a pixel in the color map\n" ) ;
                return 0 ;
            }

            image->g->SavedImages[i].RasterBits[j] = found_index ;
        }
    }


    /* Write the final image */
    if ( !output_modified_read_gif( filename, image->g ) ) { return 0 ; }

    return 1 ;
}
void apply_gray_filter( animated_gif * image ) {
    int i, j ;
    pixel ** p ;

    p = image->p ;

    for ( i = 0 ; i < image->n_images ; i++ )
    {
        for ( j = 0 ; j < image->width[i] * image->height[i] ; j++ )
        {
            int moy ;

            // moy = p[i][j].r/4 + ( p[i][j].g * 3/4 ) ;
            moy = (p[i][j].r + p[i][j].g + p[i][j].b)/3 ;
            if ( moy < 0 ) moy = 0 ;
            if ( moy > 255 ) moy = 255 ;

            p[i][j].r = moy ;
            p[i][j].g = moy ;
            p[i][j].b = moy ;
        }
    }
}
void apply_gray_line( animated_gif * image ) {
    int i, j, k ;
    pixel ** p ;

    p = image->p ;

    for ( i = 0 ; i < image->n_images ; i++ )
    {
        for ( j = 0 ; j < 10 ; j++ )
        {
            for ( k = image->width[i]/2 ; k < image->width[i] ; k++ )
            {
            p[i][CONV(j,k,image->width[i])].r = 0 ;
            p[i][CONV(j,k,image->width[i])].g = 0 ;
            p[i][CONV(j,k,image->width[i])].b = 0 ;
            }
        }
    }
}
void apply_blur_filter( animated_gif * image, int size, int threshold ) {
    int i, j, k ;
    int width, height ;
    int end = 0 ;
    int n_iter = 0 ;
    pixel ** p ;
	pixel * s_p;
    pixel * newp ;
    /* Get the pixels of all images */
    p = image->p ;

    /* Process all images */
    for ( i = 0 ; i < image->n_images ; i++ ) {
        n_iter = 0 ;
        width = image->width[i];
        height = image->height[i];

        /* Allocate array of new pixels */
        newp = (pixel *)malloc(width * height * sizeof( pixel ) ) ;

        /* Perform at least one blur iteration */
        do {
            end = 1 ;
            n_iter++ ;
			s_p = p[i];
            /* Apply blur on top part of image (10%) */
			apply_blur_top(height, width, size, s_p, newp);

            /* Copy the middle part of the image */
            for(j=height/10-size; j<height*0.9+size; j++) {
                for(k=size; k<width-size; k++) {
                    newp[CONV(j,k,width)].r = p[i][CONV(j,k,width)].r ; 
                    newp[CONV(j,k,width)].g = p[i][CONV(j,k,width)].g ; 
                    newp[CONV(j,k,width)].b = p[i][CONV(j,k,width)].b ; 
                }
            }

            /* Apply blur on the bottom part of the image (10%) */
            for(j=height*0.9+size; j<height-size; j++) {
                for(k=size; k<width-size; k++) {
                    int stencil_j, stencil_k ;
                    int t_r = 0 ;
                    int t_g = 0 ;
                    int t_b = 0 ;

                    for ( stencil_j = -size ; stencil_j <= size ; stencil_j++ ) {
                        for ( stencil_k = -size ; stencil_k <= size ; stencil_k++ ) {
                            t_r += p[i][CONV(j+stencil_j,k+stencil_k,width)].r ;
                            t_g += p[i][CONV(j+stencil_j,k+stencil_k,width)].g ;
                            t_b += p[i][CONV(j+stencil_j,k+stencil_k,width)].b ;
                        }
                    }
                    newp[CONV(j,k,width)].r = t_r / ( (2*size+1)*(2*size+1) ) ;
                    newp[CONV(j,k,width)].g = t_g / ( (2*size+1)*(2*size+1) ) ;
                    newp[CONV(j,k,width)].b = t_b / ( (2*size+1)*(2*size+1) ) ;
                }
            }

            for(j = size; j < height - size; j++) {
                for(k = size; k < width - size; k++) {
                    float diff_r ;
                    float diff_g ;
                    float diff_b ;

                    diff_r = (newp[CONV(j  ,k  ,width)].r - p[i][CONV(j  ,k  ,width)].r) ;
                    diff_g = (newp[CONV(j  ,k  ,width)].g - p[i][CONV(j  ,k  ,width)].g) ;
                    diff_b = (newp[CONV(j  ,k  ,width)].b - p[i][CONV(j  ,k  ,width)].b) ;

                    if ( diff_r > threshold || -diff_r > threshold || diff_g > threshold || -diff_g > threshold || diff_b > threshold || -diff_b > threshold) {
                        end = 0 ;
                    }

                    p[i][CONV(j  ,k  ,width)].r = newp[CONV(j  ,k  ,width)].r ;
                    p[i][CONV(j  ,k  ,width)].g = newp[CONV(j  ,k  ,width)].g ;
                    p[i][CONV(j  ,k  ,width)].b = newp[CONV(j  ,k  ,width)].b ;
                }
            }
        } while ( threshold > 0 && !end ) ;
        // printf( "Nb iter for image %d\n", n_iter ) ;
        free (newp) ;
    }

}
void apply_sobel_filter( animated_gif * image ) {
    int i, j, k ;
    int width, height ;

    pixel ** p ;

    p = image->p ;

    for ( i = 0 ; i < image->n_images ; i++ )
    {
        width = image->width[i] ;
        height = image->height[i] ;

        pixel * sobel ;

        sobel = (pixel *)malloc(width * height * sizeof( pixel ) ) ;

        for(j=1; j<height-1; j++)
        {
            for(k=1; k<width-1; k++)
            {
                int pixel_blue_no, pixel_blue_n, pixel_blue_ne;
                int pixel_blue_so, pixel_blue_s, pixel_blue_se;
                int pixel_blue_o , pixel_blue  , pixel_blue_e ;

                float deltaX_blue ;
                float deltaY_blue ;
                float val_blue;

                pixel_blue_no = p[i][CONV(j-1,k-1,width)].b ;
                pixel_blue_n  = p[i][CONV(j-1,k  ,width)].b ;
                pixel_blue_ne = p[i][CONV(j-1,k+1,width)].b ;
                pixel_blue_so = p[i][CONV(j+1,k-1,width)].b ;
                pixel_blue_s  = p[i][CONV(j+1,k  ,width)].b ;
                pixel_blue_se = p[i][CONV(j+1,k+1,width)].b ;
                pixel_blue_o  = p[i][CONV(j  ,k-1,width)].b ;
                pixel_blue    = p[i][CONV(j  ,k  ,width)].b ;
                pixel_blue_e  = p[i][CONV(j  ,k+1,width)].b ;

                deltaX_blue = -pixel_blue_no + pixel_blue_ne - 2*pixel_blue_o + 2*pixel_blue_e - pixel_blue_so + pixel_blue_se;             

                deltaY_blue = pixel_blue_se + 2*pixel_blue_s + pixel_blue_so - pixel_blue_ne - 2*pixel_blue_n - pixel_blue_no;

                val_blue = sqrt(deltaX_blue * deltaX_blue + deltaY_blue * deltaY_blue)/4;


                if ( val_blue > 50 ) 
                {
                    sobel[CONV(j  ,k  ,width)].r = 255 ;
                    sobel[CONV(j  ,k  ,width)].g = 255 ;
                    sobel[CONV(j  ,k  ,width)].b = 255 ;
                } else
                {
                    sobel[CONV(j  ,k  ,width)].r = 0 ;
                    sobel[CONV(j  ,k  ,width)].g = 0 ;
                    sobel[CONV(j  ,k  ,width)].b = 0 ;
                }
            }
        }

        for(j=1; j<height-1; j++)
        {
            for(k=1; k<width-1; k++)
            {
                p[i][CONV(j  ,k  ,width)].r = sobel[CONV(j  ,k  ,width)].r ;
                p[i][CONV(j  ,k  ,width)].g = sobel[CONV(j  ,k  ,width)].g ;
                p[i][CONV(j  ,k  ,width)].b = sobel[CONV(j  ,k  ,width)].b ;
            }
        }

        free (sobel) ;
    }

}
int main( int argc, char ** argv ) {
    char * input_filename ; 
    char * output_filename ;
    animated_gif * image ;
	#ifdef _WIN32
	LARGE_INTEGER frequency;        // ticks per second
	LARGE_INTEGER t1, t2;           // ticks
	#else
	struct timeval t1, t2;
	#endif

    double duration ;

    if ( argc < 3 ) {
        fprintf( stderr, "Usage: %s input.gif output.gif\n Only %d argument(s) were passed\n", argv[0], argc ) ;
		for (int i = 1; i < argc; i++) {
			fprintf(stderr, "Argument %d: %s\n", i,argv[i]);
		}
		return 1 ;
    }
        
    input_filename = argv[1] ;
    output_filename = argv[2] ;
		/************************
		*	IMPORT GIF IMAGE	*
		************************/
    /* IMPORT Timer start */
	#ifdef _WIN32
	// get ticks per second
	QueryPerformanceFrequency(&frequency);
	// start timer
	QueryPerformanceCounter(&t1);
	#else
	gettimeofday(&t1, NULL);
	#endif

    /* Load file and store the pixels in array */
    image = load_pixels( input_filename ) ;
    if ( image == NULL ) { return 1 ; }
    /* IMPORT Timer stop */
	#ifdef _WIN32
	QueryPerformanceFrequency(&frequency); 	// get ticks per second
	QueryPerformanceCounter(&t2); 	// start timer
	duration = (t2.QuadPart - t1.QuadPart) * 1000.0 / frequency.QuadPart; 	// compute and print the elapsed time in millisec
	#else
	gettimeofday(&t2, NULL);
	duration = (t2.tv_sec - t1.tv_sec) + ((t2.tv_usec - t1.tv_usec) / 1e6);
	#endif
    //printf( "GIF loaded from file %s with %d image(s) in %lf s\n", input_filename, image->n_images, duration );
	fprintf(stdout, "L %lf\n", duration);

		/************************
		*	APPLY GRAY FILTER	*
		************************/
    /* FILTER Timer start */
	#ifdef _WIN32
	QueryPerformanceFrequency(&frequency);	// get ticks per second
	QueryPerformanceCounter(&t1);			// start timer
	#else
	gettimeofday(&t1, NULL);
	#endif
    /* Convert the pixels into grayscale */
    apply_gray_filter( image ) ;
	/* FILTER Timer stop */
	#ifdef _WIN32
	// get ticks per second
	QueryPerformanceFrequency(&frequency);
	// start timer
	QueryPerformanceCounter(&t2);
	// compute and print the elapsed time in millisec
	duration = (t2.QuadPart - t1.QuadPart) * 1000.0 / frequency.QuadPart;
	#else
	gettimeofday(&t2, NULL);
	duration = (t2.tv_sec - t1.tv_sec) + ((t2.tv_usec - t1.tv_usec) / 1e6);
	#endif
	fprintf(stdout, "G %lf\n", duration);

		/************************
		*	APPLY BLUR FILTER	*
		************************/
	/* FILTER Timer start */
	#ifdef _WIN32
	QueryPerformanceFrequency(&frequency); 	// get ticks per second
	QueryPerformanceCounter(&t1); 	// start timer
	#else
	gettimeofday(&t1, NULL);
	#endif
    /* Apply blur filter with convergence value */
    apply_blur_filter( image, 5, 20 ) ;
	/* FILTER Timer stop */
	#ifdef _WIN32
	QueryPerformanceFrequency(&frequency); 	// get ticks per second
	QueryPerformanceCounter(&t2); 	// start timer
	duration = (t2.QuadPart - t1.QuadPart) * 1000.0 / frequency.QuadPart; 	// compute and print the elapsed time in millisec
	#else
	gettimeofday(&t2, NULL);
	duration = (t2.tv_sec - t1.tv_sec) + ((t2.tv_usec - t1.tv_usec) / 1e6);
	#endif
	fprintf(stdout, "B %lf\n", duration);

		/************************
		*	APPLY SOBEL FILTER	*
		************************/
	/* FILTER Timer start */
	#ifdef _WIN32
	QueryPerformanceFrequency(&frequency); 	// get ticks per second
	QueryPerformanceCounter(&t1); 	// start timer
	#else
	gettimeofday(&t1, NULL);
	#endif
    /* Apply sobel filter on pixels */
    apply_sobel_filter( image ) ;
    /* FILTER Timer stop */
	#ifdef _WIN32
	QueryPerformanceFrequency(&frequency); 	// get ticks per second
	QueryPerformanceCounter(&t2); 	// start timer
	duration = (t2.QuadPart - t1.QuadPart) * 1000.0 / frequency.QuadPart; 	// compute and print the elapsed time in millisec
	#else
	gettimeofday(&t2, NULL);
	duration = (t2.tv_sec - t1.tv_sec) + ((t2.tv_usec - t1.tv_usec) / 1e6);
	#endif
	fprintf(stdout, "S %lf\n", duration);
    //printf( "SOBEL done in %lf s\n", duration ) ;

		/************************
		*	EXPORT GIF IMAGE	*
		************************/
    /* EXPORT Timer start */
	#ifdef _WIN32
	QueryPerformanceFrequency(&frequency); 	// get ticks per second
	QueryPerformanceCounter(&t1); 	// start timer
	#else
	gettimeofday(&t1, NULL);
	#endif
    /* Store file from array of pixels to GIF file */
    if ( !store_pixels( output_filename, image ) ) { return 1 ; }
    /* EXPORT Timer stop */
	#ifdef _WIN32
	QueryPerformanceFrequency(&frequency); 	// get ticks per second
	QueryPerformanceCounter(&t2); 	// start timer
	duration = (t2.QuadPart - t1.QuadPart) * 1000.0 / frequency.QuadPart; 	// compute and print the elapsed time in millisec
	#else
	gettimeofday(&t2, NULL);
	duration = (t2.tv_sec - t1.tv_sec) + ((t2.tv_usec - t1.tv_usec) / 1e6);
	#endif
	fprintf(stdout, "E %lf\n", duration);
    //printf( "Export done in %lf s in file %s\n", duration, output_filename ) ;

    return 0 ;
}
